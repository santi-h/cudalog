#include "hip/hip_runtime.h"
#include "hostlog.h"
#include "commonlog.h"
#include <assert.h>
#include <process.h>    /* _beginthread, _endthread */
#include <stdio.h>      /* FILE, fopen, fclose, ... */
#include <windows.h>    /* HANDLE, WaitForSingleObject, CloseHandle, ... */
#include <string.h>

__device__ logform* dforms = 0;
logform* hforms = 0;

struct logsession
{
    bool running;
    int nthreads;
    HANDLE threadHandle;
    FILE* file;
    FILE** files;
} session;

unsigned __stdcall thread( void* in)
{
    for( int i=0; session.running; i = (i+1)%session.nthreads)
    {
        if( *hforms[i].drain)
        {
            FILE* file = session.file ? session.file : session.files[i];
            fwrite(hforms[i].buffer, 1, *hforms[i].written, file);
            *hforms[i].drain = 0;
        }
    }
    _endthreadex( 0 );
    return 0;
}

void cudaStartLog(int blocks, int threadsPerBlock, const char* opt)
{
    cudaStartLog( dim3(blocks), dim3(threadsPerBlock), opt);
}

void cudaStartLog(const dim3& grid, const dim3& block, const char* opt)
{
    int nthreads = (grid.x*grid.y*grid.z)*(block.x*block.y*block.z);
    assert( nthreads > 0);
    size_t forms_size = sizeof(logform) * nthreads;
    logform* h_dforms = 0;

    h_dforms = (logform*)malloc( forms_size);
    hforms = (logform*)malloc( forms_size);

    if( opt && strcmp( opt, "SEPARATE") == 0)
    {
        session.files = (FILE**)malloc(sizeof(FILE*)*nthreads);
        session.file = 0;
    }
    else
    {
        session.files = 0;
        session.file = fopen("log.txt", "wb");
    }    

    for( int i=0; i<nthreads; i++)
    {
        assert( hipHostAlloc(&hforms[i].drain, sizeof(bool), hipHostMallocMapped) == hipSuccess);
        assert( hipHostGetDevicePointer(&h_dforms[i].drain, hforms[i].drain, 0) == hipSuccess);
        
        assert( hipHostAlloc(&hforms[i].buffer, sizeof(uint8_t)*LOG_BUFFER_SIZE, hipHostMallocMapped) == hipSuccess);
        assert( hipHostGetDevicePointer(&h_dforms[i].buffer, hforms[i].buffer, 0) == hipSuccess);

        assert( hipHostAlloc(&hforms[i].written, sizeof(size_t), hipHostMallocMapped) == hipSuccess);
        assert( hipHostGetDevicePointer(&h_dforms[i].written, hforms[i].written, 0) == hipSuccess);
        
        if( session.files)
        {
            char tid[80];
            char filename[80] = "log";
            sprintf(tid, "%d", i);
            strcat( strcat( filename, tid), ".txt");
            session.files[i] = fopen(filename, "wb");
        }
    }

    logform* d_dforms = 0;
    assert( hipMalloc(&d_dforms, forms_size) == hipSuccess);
    assert( hipMemcpy(d_dforms, h_dforms, forms_size, hipMemcpyHostToDevice) == hipSuccess);
    assert( hipMemcpyToSymbol(HIP_SYMBOL(dforms), &d_dforms, sizeof(logform*)) == hipSuccess);

    free( h_dforms);

    session.running = 1;
    session.nthreads = nthreads;
    session.threadHandle = (HANDLE)_beginthreadex( NULL, 0, thread, NULL, 0, NULL);
    assert( session.threadHandle);
}

void cudaStopLog()
{
    // make sure thread finishes
    session.running = 0;
    WaitForSingleObject( session.threadHandle, INFINITE );
    CloseHandle( session.threadHandle);

    // deallocate device memory for forms
    logform* d_dforms = 0;
    assert( hipMemcpyFromSymbol(&d_dforms, HIP_SYMBOL(dforms), sizeof( logform*)) == hipSuccess);
    assert( hipFree( d_dforms) == hipSuccess);
    d_dforms = 0;
    assert( hipMemcpyToSymbol(HIP_SYMBOL(dforms), &d_dforms, sizeof(logform*)) == hipSuccess);

    // deallocate each form field, write last logs
    for( int i=0; i<session.nthreads; i++)
    {
        if( *hforms[i].written > 0)
        {
            FILE* file = session.file ? session.file : session.files[i];
            fwrite(hforms[i].buffer, 1, *hforms[i].written, file);
        }
        assert( hipHostFree( hforms[i].drain) == hipSuccess);
        assert( hipHostFree( hforms[i].buffer) == hipSuccess);
        assert( hipHostFree( hforms[i].written) == hipSuccess);
        if( session.files) fclose( session.files[i]);
    }
    free( hforms);
    hforms = 0;
    if( session.files) free( session.files);
    if( session.file) fclose( session.file);
    session.file = 0;
}
